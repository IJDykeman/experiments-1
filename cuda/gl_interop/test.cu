#include "hip/hip_runtime.h"
#ifndef _DIST_KERNEL_CU_
#define _DIST_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

__device__ uint rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) |
         (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void d_render(uint *d_output, uint imageW, uint imageH, float scale,
                         float2 view_center) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= imageW) || (y >= imageH)) {
    return;
  }

  // These int-float cast operations aren't cheap
  float u = ((x / (float)imageW) * 2.0f - 1.0f);  // + view_center.x;
  float v = ((y / (float)imageH) * 2.0f - 1.0f);  // + view_center.y;

  float4 sum = make_float4(0.0f);
  float distance = hypot(u, v);
  sum.x = distance;
  sum.y = distance / sqrt(distance) * v;
  sum.z = distance * distance;
  sum.w = 1.0;

  d_output[y * imageW + x] = rgbaFloatToInt(sum);
}

extern "C" void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output,
                              uint imageW, uint imageH, float scale,
                              float2 view_center) {
  printf("%9.2f, %9.2f: %9.2f \n", view_center.x, view_center.y, scale);
  d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, scale,
                                    view_center);
}

#endif  // #ifndef _DIST_KERNEL_CU_
